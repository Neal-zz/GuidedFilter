#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"
#include "guidedFilter.cuh"
#include "config.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <vector>
#include <thread>

// test function.
void darkChannel_cpu(std::vector<unsigned char> pixels, std::vector<unsigned char> darkImage, unsigned width, unsigned height) {
	int loss = 0;
	for (int x = 0; x < width; x++) {
		for (int y = 0; y < height; y++) {
			int value = 255;
			for (int dx = -windowR; dx <= windowR; dx++) {
				for (int dy = -windowR; dy <= windowR; dy++) {
					for (int ch = 0; ch < 3; ch++) {
						int tempx = x + dx;
						int tempy = y + dy;
						tempx = tempx > 0 ? tempx : 0;
						tempy = tempy > 0 ? tempy : 0;
						tempx = tempx < (width - 1) ? tempx : (width - 1);
						tempy = tempy < (height - 1) ? tempy : (height - 1);

						int tempv = static_cast<int>(pixels[tempy * width * 4 + tempx * 4 + ch]);
						if (tempv < value) {
							value = tempv;
						}
					}
				}
			}
			loss += abs(value - static_cast<int>(darkImage[y * width + x]));
		}
	}
	std::cout << "loss: " << loss << std::endl;

	return;
}

/*
Class to calculate time taken by functions in seconds.
* Creating an object of the class in a function, calls the constructor which starts the timer.
* At the end of the function, the destructor is called which stops the timer and calculates the duration.
* We can get the duration manually using the getElapsedTime method.
*/
class Timer {
private:
	std::chrono::time_point<std::chrono::steady_clock> m_Start, m_End;
	std::chrono::duration<float> m_Duration;

public:
	Timer() {
		m_Start = std::chrono::high_resolution_clock::now();
	}

	~Timer() {
		m_End = std::chrono::high_resolution_clock::now();
		m_Duration = m_End - m_Start;

		std::cout << "Done (" << m_Duration.count() << " s)" << std::endl;
	}

	float getElapsedTime() {
		m_End = std::chrono::high_resolution_clock::now();
		m_Duration = m_End - m_Start;

		return m_Duration.count();
	}
};


// Display GPU info
// https://stackoverflow.com/a/5689133
void DisplayHeader() {
	const int kb = 1024;
	const int mb = kb * kb;
	std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

	std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	std::cout << "CUDA Devices: " << std::endl << std::endl;

	for (int i = 0; i < devCount; ++i) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
		std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << std::endl;
		std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << std::endl;
		std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" << std::endl;
		std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

		std::cout << "  Warp size:         " << props.warpSize << std::endl;
		std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
		std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
		std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << std::endl;
		std::cout << std::endl;
	}
}

std::vector<unsigned char> loadImage(const char* filename, unsigned& width, unsigned& height) {
	Timer timer;

	std::vector<unsigned char> pixels;

	unsigned error = lodepng::decode(pixels, width, height, filename);
	if (error) {
		std::cout << "Failed to load image: " << lodepng_error_text(error) << std::endl;
		std::cin.get();
		exit(-1);
	}

	return pixels;
}

void CudaCall(const hipError_t& status) {
	if (status != hipSuccess) {
		std::cout << "Error [" << status << "]: " << hipGetErrorString(status) << " (" << __FILE__ << ":" << __LINE__ << ")" << std::endl;
	}
}

std::vector<unsigned char> showResult(const std::vector<unsigned char>& in,
	const unsigned width, const unsigned height) {
	std::vector<unsigned char> result(width * height * 4);

	for (int i = 0; i < width * height * 4; i += 4) {
		float temp = in[i / 4];

		result[i] = result[i + 1] = result[i + 2] = static_cast<unsigned char>(temp);
		result[i + 3] = 255;
	}

	return result;
}

int main()
{
	Timer timer;

	DisplayHeader();

	// read image
	std::vector<unsigned char> pixels;  // 1 byte: 0-255
	unsigned width, height;
	std::cout << "Reading Fog Image...";
	pixels = loadImage("fog0.png", width, height);
	unsigned imSize = width * height;

	// block and thread allocation.
	dim3 blocks((width + tileWidth - 1) / tileWidth, (height + tileHeight - 1) / tileHeight);
	dim3 threads_withAprone(aproneWidth, aproneHeight);  // waste threads.
	dim3 threads_noAprone(tileWidth, tileHeight);

	dim3 threads_subHist(512);
	dim3 blocks_subHist((width + threads_subHist.x - 1) / threads_subHist.x);
	//unsigned nsubHist = blocks.x * blocks.y;  // for subHist.
	unsigned nsubHist = blocks_subHist.x;  // for subHist_2.
	unsigned subHistSize = nsubHist * nbins;

	dim3 threads_sumHist(nbins);
	dim3 blocks_sumHist(1);

	dim3 threads_getAc(3);
	dim3 blocks_getAc(1);

	std::vector<unsigned char> output(imSize);
	std::vector<unsigned> h_hist(nbins);

	// Device variabels
	unsigned char* d_orig, * d_Idark, * d_AcRow, * d_Ac, * d_guidedI;
	unsigned* d_subHist, *d_hist;
	float* d_ttilde, * d_ttilde2, * d_ab, * d_outputQ;

	CudaCall(hipMalloc((void**)&d_orig, sizeof(unsigned char) * imSize * 4));  // 4: rgbd.
	CudaCall(hipMalloc((void**)&d_Idark, sizeof(unsigned char) * imSize));
	CudaCall(hipMalloc((void**)&d_AcRow, sizeof(unsigned char) * width * 3));
	CudaCall(hipMalloc((void**)&d_Ac, sizeof(unsigned char) * 3));  // 3: rgb
	CudaCall(hipMalloc((void**)&d_ttilde, sizeof(float) * imSize));
	CudaCall(hipMalloc((void**)&d_ttilde2, sizeof(float) * imSize));
	CudaCall(hipMalloc((void**)&d_guidedI, sizeof(unsigned char) * imSize));
	CudaCall(hipMalloc((void**)&d_subHist, sizeof(unsigned) * subHistSize));
	CudaCall(hipMalloc((void**)&d_hist, sizeof(unsigned) * nbins));
	CudaCall(hipMalloc((void**)&d_ab, sizeof(float) * imSize * 2));  // 2: ab.
	CudaCall(hipMalloc((void**)&d_outputQ, sizeof(float) * imSize));

	// Copy Data from host to device
	CudaCall(hipMemcpy(d_orig, pixels.data(), sizeof(pixels[0]) * pixels.size(), hipMemcpyHostToDevice));
	// Profiling
	float elapsed = 0;
	hipEvent_t start, stop;

	CudaCall(hipEventCreate(&start));
	CudaCall(hipEventCreate(&stop));
 
	// I dark.
	std::cout << "Get the Dark Image of I...";
	CudaCall(hipEventRecord(start));

	darkChannel<<<blocks, threads_noAprone>>>(d_orig, d_Idark, width, height);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	//std::vector<unsigned char> darkImage_test(imSize);
	//CudaCall(hipMemcpy(&darkImage_test[0], d_Idark, sizeof(darkImage_test[0]) * imSize, hipMemcpyDeviceToHost));
	//darkChannel_cpu(pixels, darkImage_test, width, height);

	// subhistogram.
	std::cout << "Calculate the Subhistogram...";
	CudaCall(hipEventRecord(start));

	//subHist<<<blocks, threads_noAprone>>>(d_Idark, d_subHist, width, height);  // slower.
	subHist_2<<<blocks_subHist, threads_subHist>>>(d_Idark, d_subHist, width, height);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// sum subhistograms.
	std::cout << "Sum Subhistograms...";
	CudaCall(hipEventRecord(start));

	sumHist<<<blocks_sumHist, threads_sumHist>>>(d_subHist, d_hist, subHistSize);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// get AcRow.
	std::cout << "Calculate AcRow...";
	CudaCall(hipEventRecord(start));

	unsigned char colorThresh = 0;
	unsigned totalNum = 0;
	CudaCall(hipMemcpy(&h_hist[0], d_hist, sizeof(h_hist[0]) * nbins, hipMemcpyDeviceToHost));
	for (int bin = (nbins - 1); bin > -1; bin--) {
		totalNum += h_hist[bin];
		if (totalNum >= (imSize * 0.001)) {
			colorThresh = static_cast<unsigned char>(bin);
			break;
		}
	}
	getAcRow<<<blocks_subHist, threads_subHist>>>(d_orig, d_Idark, d_AcRow, colorThresh, width, height);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// get Ac.
	std::cout << "Calculate Ac...";
	CudaCall(hipEventRecord(start));

	getAc<<<blocks_getAc, threads_getAc>>>(d_AcRow, d_Ac, width);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// get t_tilde.
	std::cout << "Get the t_tilde(x)...";
	CudaCall(hipEventRecord(start));

	getttilde<<<blocks, threads_withAprone>>>(d_orig, d_ttilde, d_Ac, width, height);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Scale and gray guided I.
	std::cout << "Converting Guided I to Grayscale...";
	CudaCall(hipEventRecord(start));

	ScaleAndGray<<<height, width>>>(d_orig, d_guidedI, width, height, scaleFactor);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// calculate ak and bk for each window wk.
	std::cout << "Calculating ak and bk...";
	CudaCall(hipEventRecord(start));

	linearPara<<<blocks, threads_withAprone>>>(d_ttilde, d_guidedI, d_ab, width, height, epsilon);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// image filtering
	std::cout << "Image Filtering...";
	CudaCall(hipEventRecord(start));

	doFiltering_new<<<blocks, threads_withAprone>>>(d_ab, d_guidedI, d_ttilde2, width, height);

	CudaCall(hipEventRecord(stop));
	CudaCall(hipEventSynchronize(stop));
	CudaCall(hipEventElapsedTime(&elapsed, start, stop));
	std::cout << "Done (" << elapsed / 1000 << " s)" << std::endl;

	CudaCall(hipPeekAtLastError());
	CudaCall(hipDeviceSynchronize());

	// Copy data from device to host
	CudaCall(hipMemcpy(&output[0], d_Idark, sizeof(output[0]) * imSize, hipMemcpyDeviceToHost));
	
	lodepng::encode("output.png", showResult(output, width, height), width, height);  // draw right image on the left image.
	
	//unsigned num = 0;
	//for (int i = 0; i < nbins; i++) {
	//	num += h_hist[i];
	//}
	//std::cout << num << std::endl;

	//std::cout << static_cast<int>(output[0]) << " " << static_cast<int>(output[1]) << " " << static_cast<int>(output[2]) << " " << std::endl;

	//std::cout << "The program took " << timer.getElapsedTime() << " s" << std::endl;

	hipFree(d_orig);
	hipFree(d_Idark);
	hipFree(d_AcRow);
	hipFree(d_Ac);
	hipFree(d_ttilde);
	hipFree(d_ttilde2);
	hipFree(d_guidedI);
	hipFree(d_subHist);
	hipFree(d_hist);
	hipFree(d_ab);
	hipFree(d_outputQ);

	std::cin.get();

    return 0;
}


